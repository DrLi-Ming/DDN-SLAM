/**
* This file is part of ORB-SLAM2.
*
* Copyright (C) 2014-2016 Raúl Mur-Artal <raulmur at unizar dot es> (University of Zaragoza)
* For more information see <https://github.com/raulmur/ORB_SLAM2>
*
* ORB-SLAM2 is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* ORB-SLAM2 is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with ORB-SLAM2. If not, see <http://www.gnu.org/licenses/>.
*/


#include<iostream>
#include<algorithm>
#include<fstream>
#include<chrono>
#include <sys/wait.h>
#include<unistd.h>

#include<opencv2/core/core.hpp>
#include<opencv2/imgcodecs/legacy/constants_c.h>

#include<System.h>

// #define COMPILEDWITHC11 // Hack through this
using namespace std;

void LoadImages(const string &strFile, vector<string> &vstrImageFilenames,
                vector<double> &vTimestamps);
string GetDatasetName(const string &strSequencePath);

int main(int argc, char **argv)
{
    if(argc != 4)
    {
        cerr << endl << "Usage: ./mono_tum path_to_vocabulary path_to_settings path_to_sequence" << endl;
        return 1;
    }

    // Retrieve paths to images
    vector<string> vstrImageFilenames;
    vector<double> vTimestamps;
    string strFile = string(argv[3])+"/rgb.txt";
    LoadImages(strFile, vstrImageFilenames, vTimestamps);

    int nImages = vstrImageFilenames.size();

    // Create SLAM system. It initializes all system threads and gets ready to process frames.
    ORB_SLAM3::System SLAM(argv[1],argv[2],ORB_SLAM3::System::MONOCULAR,true);

    // Vector for tracking time statistics
    vector<float> vTimesTrack;
    vTimesTrack.resize(nImages);

    cout << endl << "-------" << endl;
    cout << "Start processing sequence ..." << endl;
    cout << "Images in the sequence: " << nImages << endl << endl;

    // Main loop
    cv::Mat im;
    for(int ni=0; ni<nImages; ni++)
    {
        // Read image from file
        im = cv::imread(string(argv[3])+"/"+vstrImageFilenames[ni],CV_LOAD_IMAGE_UNCHANGED);
        double tframe = vTimestamps[ni];

        if(im.empty())
        {
            cerr << endl << "Failed to load image at: "
                 << string(argv[3]) << "/" << vstrImageFilenames[ni] << endl;
            return 1;
        }


        std::chrono::steady_clock::time_point t1 = std::chrono::steady_clock::now();

        // Pass the image to the SLAM system
        SLAM.TrackMonocular(im,tframe);

        std::chrono::steady_clock::time_point t2 = std::chrono::steady_clock::now();

        double ttrack= std::chrono::duration_cast<std::chrono::duration<double> >(t2 - t1).count();
        vTimesTrack[ni]=ttrack;

        // Wait to load the next frame
        double T=0;
        if(ni<nImages-1)
            T = vTimestamps[ni+1]-tframe;
        else if(ni>0)
            T = tframe-vTimestamps[ni-1];

        if(ttrack<T)
            usleep((T-ttrack)*1e6);
    }

    // Stop orb-viewer and tracking. 
    // The user can watch the Nerf screen
    SLAM.Spin();

    // Tracking time statistics
    sort(vTimesTrack.begin(),vTimesTrack.end());
    float totaltime = 0;
    for(int ni=0; ni<nImages; ni++)
    {
        totaltime+=vTimesTrack[ni];
    }
    cout << "-------" << endl << endl;
    cout << "median tracking time: " << vTimesTrack[nImages/2] << endl;
    cout << "mean tracking time: " << totaltime/nImages << endl;

    string dataset_name = GetDatasetName(string(argv[3])); 
    auto trajString = "evaluation/MONO_TUM_"+dataset_name+"_KeyFrameTrajectory";
    auto snapString = "evaluation/MONO_TUM_"+dataset_name+".msgpack";
    auto gtJsonTrajString = "evaluation/MONO_TUM_"+dataset_name+"_gtTraj.json";

    // Save camera trajectory
    SLAM.SaveKeyFrameTrajectoryTUM(trajString+".txt");  // rpj only
    SLAM.SaveKeyFrameTrajectoryNGP(trajString+".json"); // rpj (+ pht if train extrinsics) 
    SLAM.SaveSnapShot(snapString);

    int pid = fork();
    if (pid < 0)
    {
        cout << "fork failed" << endl;
    }
    else if (pid == 0)
    {
        // For headless version, we do not need to spin the program.
        // But instead, terminate training process and execute evaluation script.
        auto gtString = string(argv[3]) + "/groundtruth.txt";
        auto trajPathString = trajString + ".txt";
        auto plotString = trajString + ".png";
        char *gtPath = (char *)(gtString.c_str());
        char *trajPath = (char *)(trajPathString.c_str());
        char *plotPath = (char *)(plotString.c_str());
        char *gtJsonTrajPath = (char *)(gtJsonTrajString.c_str());

        std::cout << "ATE w/ reprojection error:" << std::endl;
        char *execArgs[] = {"python3", "scripts/evaluate_ate.py", gtPath, trajPath, "--verbose", "--plot", plotPath, "--save_gt_json", gtJsonTrajPath, NULL};
        execvp("python3", execArgs);
    }
    wait(NULL);
    
    std::cout << std::endl;

    pid = fork();
    if (pid < 0)
    {
        cout << "fork failed" << endl;
    }
    else if( pid == 0 )
    {
        // For headless version, we do not need to spin the program.
        // But instead, terminate training process and execute evaluation script.
        auto gtString = string(argv[3]) + "/groundtruth.txt";
        auto trajPathString = trajString + ".json";
        auto plotString = trajString + "_rpj+pht.png";
        char *gtPath = (char *)(gtString.c_str());
        char *trajPath = (char *)(trajPathString.c_str());
        char *plotPath = (char *)(plotString.c_str());

        std::cout << "ATE w/ reprojection error (+ photometric error if optimize extrinsic == true):" << std::endl;
        char *execArgs[] = {"python3", "scripts/evaluate_ate.py", gtPath, trajPath, "--verbose", "--plot", plotPath, NULL};
        execvp("python3", execArgs);
    }
    wait(NULL);

#ifdef ORBEEZ_GUI
    cout << "Press ctrl + c to exit the program " << endl;

    // Don't stop program, to see the Nerf training result
    volatile int keep_spinning = 1;
    while (keep_spinning) ; // spin
#endif  

    return 0;
}

void LoadImages(const string &strFile, vector<string> &vstrImageFilenames, vector<double> &vTimestamps)
{
    ifstream f;
    f.open(strFile.c_str());

    // skip first three lines
    string s0;
    getline(f,s0);
    getline(f,s0);
    getline(f,s0);

    while(!f.eof())
    {
        string s;
        getline(f,s);
        if(!s.empty())
        {
            stringstream ss;
            ss << s;
            double t;
            string sRGB;
            ss >> t;
            vTimestamps.push_back(t);
            ss >> sRGB;
            vstrImageFilenames.push_back(sRGB);
        }
    }
}

string GetDatasetName(const string &strSequencePath) 
{
    string s(strSequencePath);
    std::string delimiter = "/";

    size_t pos = 0;
    std::string token;
    while ((pos = s.find(delimiter)) != std::string::npos) {
        token = s.substr(0, pos);
        s.erase(0, pos + delimiter.length());
    }

    if (s.length() == 0)
        return token;
    else
        return s;
}