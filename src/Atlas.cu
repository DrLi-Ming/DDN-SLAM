#include "hip/hip_runtime.h"
/**
* This file is part of ORB-SLAM3
*
* Copyright (C) 2017-2021 Carlos Campos, Richard Elvira, Juan J. Gómez Rodríguez, José M.M. Montiel and Juan D. Tardós, University of Zaragoza.
* Copyright (C) 2014-2016 Raúl Mur-Artal, José M.M. Montiel and Juan D. Tardós, University of Zaragoza.
*
* ORB-SLAM3 is free software: you can redistribute it and/or modify it under the terms of the GNU General Public
* License as published by the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* ORB-SLAM3 is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without even
* the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License along with ORB-SLAM3.
* If not, see <http://www.gnu.org/licenses/>.
*/

#include "Atlas.h"
#include "Viewer.h"

#include "GeometricCamera.h"
#include "Pinhole.h"
#include "KannalaBrandt8.h"

namespace ORB_SLAM3
{

Atlas::Atlas(){
    mpCurrentMap = static_cast<Map*>(NULL);
}

Atlas::Atlas(int initKFid): mnLastInitKFidMap(initKFid), mHasViewer(false)
{
    mpCurrentMap = static_cast<Map*>(NULL);
    CreateNewMap();
}

Atlas::~Atlas()
{
    for(std::set<Map*>::iterator it = mspMaps.begin(), end = mspMaps.end(); it != end;)
    {
        Map* pMi = *it;

        if(pMi)
        {
            delete pMi;
            pMi = static_cast<Map*>(NULL);

            it = mspMaps.erase(it);
        }
        else
            ++it;

    }
}

void Atlas::ORBEEZMAP(const string &strSettingPath, const string &strSlamTransform, const bool bTrainCameraWithPhotometric)
{
    mpCurrentMap->mpTestbed = std::make_shared<ngp::Testbed>(ngp::ETestbedMode::NerfSlam);
    cv::FileStorage fSettings(strSettingPath, cv::FileStorage::READ);

    // Must exist
    cv::FileNode camera_node = fSettings["Camera"];
    float fl_x                        = mpCurrentMap->KeyCheck(camera_node, "fx");
    float fl_y                        = mpCurrentMap->KeyCheck(camera_node, "fy");
    float k1                          = mpCurrentMap->KeyCheck(camera_node, "k1");
    float k2                          = mpCurrentMap->KeyCheck(camera_node, "k2");
    float p1                          = mpCurrentMap->KeyCheck(camera_node, "p1");
    float p2                          = mpCurrentMap->KeyCheck(camera_node, "p2");
    float cx                          = mpCurrentMap->KeyCheck(camera_node, "cx");
    float cy                          = mpCurrentMap->KeyCheck(camera_node, "cy");
    int width                         = mpCurrentMap->KeyCheck(camera_node, "width");
    int height                        = mpCurrentMap->KeyCheck(camera_node, "height");
    cv::FileNode NeRF_node   = fSettings["NeRF"];
    int aabb_scale                    = mpCurrentMap->KeyCheck(NeRF_node, "aabb_scale");
    float scale                       = mpCurrentMap->KeyCheck(NeRF_node, "scale");
    cv::FileNode offset               = mpCurrentMap->KeyCheck(NeRF_node, "offset");
    std::string network_config_path   = mpCurrentMap->KeyCheck(NeRF_node, "network_config_path");

    std::vector<float> offset_vector;
    for(cv::FileNodeIterator it = offset.begin(); it != offset.end(); it++)
    {
        offset_vector.push_back((float)*it);
    }
    // nlohmann::json offset_array = nlohmann::json::parse(offset_vector.begin(), offset_vector.end());
    
    // Only if RGB-D
    cv::FileNode depth_node           = fSettings["DepthMapFactor"];

    if (!depth_node.empty())
    {
        float DepthMapFactor = fSettings["DepthMapFactor"];
        // Although is name as integer, instant-ngp requires float.
        mpCurrentMap->m_scene_config["integer_depth_scale"] = 1.0f/DepthMapFactor;
    }

    mpCurrentMap->m_scene_config["fl_x"]                  = fl_x;
    mpCurrentMap->m_scene_config["fl_y"]                  = fl_y;
    mpCurrentMap->m_scene_config["k1"]                    = k1;
    mpCurrentMap->m_scene_config["k2"]                    = k2;
    mpCurrentMap->m_scene_config["p1"]                    = p1;
    mpCurrentMap->m_scene_config["p2"]                    = p2;
    mpCurrentMap->m_scene_config["cx"]                    = cx;
    mpCurrentMap->m_scene_config["cy"]                    = cy;
    mpCurrentMap->m_scene_config["w"]                     = width;
    mpCurrentMap->m_scene_config["h"]                     = height;
    mpCurrentMap->m_scene_config["aabb_scale"]            = aabb_scale;
    mpCurrentMap->m_scene_config["scale"]                 = scale;
    mpCurrentMap->m_scene_config["offset"]                = offset_vector;

    std::string json_string =  mpCurrentMap->m_scene_config.dump(4);
    std::cout << json_string << std::endl;

    std::ofstream ofs(strSlamTransform, std::ofstream::trunc);
    ofs << json_string << std::endl;
    ofs.close();

    mpCurrentMap->mpTestbed->load_training_data(strSlamTransform);
    mpCurrentMap->mpTestbed->reload_network_from_file(network_config_path);
    mpCurrentMap->mpTestbed->m_train = true;
    mpCurrentMap->mpTestbed->m_nerf.training.optimize_extrinsics = bTrainCameraWithPhotometric;
}

void Atlas::CreateNewMap()
{
    unique_lock<mutex> lock(mMutexAtlas);
    cout << "Creation of new map with id: " << Map::nNextId << endl;
    if(mpCurrentMap){
        if(!mspMaps.empty() && mnLastInitKFidMap < mpCurrentMap->GetMaxKFid())
            mnLastInitKFidMap = mpCurrentMap->GetMaxKFid()+1; //The init KF is the next of current maximum

        mpCurrentMap->SetStoredMap();
        cout << "Stored map with ID: " << mpCurrentMap->GetId() << endl;

        //if(mHasViewer)
        //    mpViewer->AddMapToCreateThumbnail(mpCurrentMap);
    }
    cout << "Creation of new map with last KF id: " << mnLastInitKFidMap << endl;

    mpCurrentMap = new Map(mnLastInitKFidMap);
    mpCurrentMap->SetCurrentMap();
    mspMaps.insert(mpCurrentMap);
}

void Atlas::ChangeMap(Map* pMap)
{
    unique_lock<mutex> lock(mMutexAtlas);
    cout << "Change to map with id: " << pMap->GetId() << endl;
    if(mpCurrentMap){
        mpCurrentMap->SetStoredMap();
    }

    mpCurrentMap = pMap;
    mpCurrentMap->SetCurrentMap();
}

unsigned long int Atlas::GetLastInitKFid()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mnLastInitKFidMap;
}

void Atlas::SetViewer(Viewer* pViewer)
{
    mpViewer = pViewer;
    mHasViewer = true;
}

void Atlas::AddKeyFrame(KeyFrame* pKF)
{
    Map* pMapKF = pKF->GetMap();
    pMapKF->AddKeyFrame(pKF);
}

void Atlas::AddMapPoint(MapPoint* pMP)
{
    Map* pMapMP = pMP->GetMap();
    pMapMP->AddMapPoint(pMP);
}

GeometricCamera* Atlas::AddCamera(GeometricCamera* pCam)
{
    //Check if the camera already exists
    bool bAlreadyInMap = false;
    int index_cam = -1;
    for(size_t i=0; i < mvpCameras.size(); ++i)
    {
        GeometricCamera* pCam_i = mvpCameras[i];
        if(!pCam) std::cout << "Not pCam" << std::endl;
        if(!pCam_i) std::cout << "Not pCam_i" << std::endl;
        if(pCam->GetType() != pCam_i->GetType())
            continue;

        if(pCam->GetType() == GeometricCamera::CAM_PINHOLE)
        {
            if(((Pinhole*)pCam_i)->IsEqual(pCam))
            {
                bAlreadyInMap = true;
                index_cam = i;
            }
        }
        else if(pCam->GetType() == GeometricCamera::CAM_FISHEYE)
        {
            if(((KannalaBrandt8*)pCam_i)->IsEqual(pCam))
            {
                bAlreadyInMap = true;
                index_cam = i;
            }
        }
    }

    if(bAlreadyInMap)
    {
        return mvpCameras[index_cam];
    }
    else{
        mvpCameras.push_back(pCam);
        return pCam;
    }
}

std::vector<GeometricCamera*> Atlas::GetAllCameras()
{
    return mvpCameras;
}

void Atlas::SetReferenceMapPoints(const std::vector<MapPoint*> &vpMPs)
{
    unique_lock<mutex> lock(mMutexAtlas);
    mpCurrentMap->SetReferenceMapPoints(vpMPs);
}

void Atlas::InformNewBigChange()
{
    unique_lock<mutex> lock(mMutexAtlas);
    mpCurrentMap->InformNewBigChange();
}

int Atlas::GetLastBigChangeIdx()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->GetLastBigChangeIdx();
}

long unsigned int Atlas::MapPointsInMap()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->MapPointsInMap();
}

long unsigned Atlas::KeyFramesInMap()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->KeyFramesInMap();
}

std::vector<KeyFrame*> Atlas::GetAllKeyFrames()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->GetAllKeyFrames();
}

std::vector<MapPoint*> Atlas::GetAllMapPoints()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->GetAllMapPoints();
}

std::vector<MapPoint*> Atlas::GetReferenceMapPoints()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->GetReferenceMapPoints();
}

vector<Map*> Atlas::GetAllMaps()
{
    unique_lock<mutex> lock(mMutexAtlas);
    struct compFunctor
    {
        inline bool operator()(Map* elem1 ,Map* elem2)
        {
            return elem1->GetId() < elem2->GetId();
        }
    };
    vector<Map*> vMaps(mspMaps.begin(),mspMaps.end());
    sort(vMaps.begin(), vMaps.end(), compFunctor());
    return vMaps;
}

int Atlas::CountMaps()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mspMaps.size();
}

void Atlas::clearMap()
{
    unique_lock<mutex> lock(mMutexAtlas);
    mpCurrentMap->clear();
}

void Atlas::clearAtlas()
{
    unique_lock<mutex> lock(mMutexAtlas);
    /*for(std::set<Map*>::iterator it=mspMaps.begin(), send=mspMaps.end(); it!=send; it++)
    {
        (*it)->clear();
        delete *it;
    }*/
    mspMaps.clear();
    mpCurrentMap = static_cast<Map*>(NULL);
    mnLastInitKFidMap = 0;
}

Map* Atlas::GetCurrentMap()
{
    unique_lock<mutex> lock(mMutexAtlas);
    if(!mpCurrentMap)
        CreateNewMap();
    while(mpCurrentMap->IsBad())
        usleep(3000);

    return mpCurrentMap;
}

void Atlas::SetMapBad(Map* pMap)
{
    mspMaps.erase(pMap);
    pMap->SetBad();

    mspBadMaps.insert(pMap);
}

void Atlas::RemoveBadMaps()
{
    /*for(Map* pMap : mspBadMaps)
    {
        delete pMap;
        pMap = static_cast<Map*>(NULL);
    }*/
    mspBadMaps.clear();
}

bool Atlas::isInertial()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->IsInertial();
}

void Atlas::SetInertialSensor()
{
    unique_lock<mutex> lock(mMutexAtlas);
    mpCurrentMap->SetInertialSensor();
}

void Atlas::SetImuInitialized()
{
    unique_lock<mutex> lock(mMutexAtlas);
    mpCurrentMap->SetImuInitialized();
}

bool Atlas::isImuInitialized()
{
    unique_lock<mutex> lock(mMutexAtlas);
    return mpCurrentMap->isImuInitialized();
}

void Atlas::PreSave()
{
    if(mpCurrentMap){
        if(!mspMaps.empty() && mnLastInitKFidMap < mpCurrentMap->GetMaxKFid())
            mnLastInitKFidMap = mpCurrentMap->GetMaxKFid()+1; //The init KF is the next of current maximum
    }

    struct compFunctor
    {
        inline bool operator()(Map* elem1 ,Map* elem2)
        {
            return elem1->GetId() < elem2->GetId();
        }
    };
    std::copy(mspMaps.begin(), mspMaps.end(), std::back_inserter(mvpBackupMaps));
    sort(mvpBackupMaps.begin(), mvpBackupMaps.end(), compFunctor());

    std::set<GeometricCamera*> spCams(mvpCameras.begin(), mvpCameras.end());
    for(Map* pMi : mvpBackupMaps)
    {
        if(!pMi || pMi->IsBad())
            continue;

        if(pMi->GetAllKeyFrames().size() == 0) {
            // Empty map, erase before of save it.
            SetMapBad(pMi);
            continue;
        }
        pMi->PreSave(spCams);
    }
    RemoveBadMaps();
}

void Atlas::PostLoad()
{
    map<unsigned int,GeometricCamera*> mpCams;
    for(GeometricCamera* pCam : mvpCameras)
    {
        mpCams[pCam->GetId()] = pCam;
    }

    mspMaps.clear();
    unsigned long int numKF = 0, numMP = 0;
    for(Map* pMi : mvpBackupMaps)
    {
        mspMaps.insert(pMi);
        pMi->PostLoad(mpKeyFrameDB, mpORBVocabulary, mpCams);
        numKF += pMi->GetAllKeyFrames().size();
        numMP += pMi->GetAllMapPoints().size();
    }
    mvpBackupMaps.clear();
}

void Atlas::SetKeyFrameDababase(KeyFrameDatabase* pKFDB)
{
    mpKeyFrameDB = pKFDB;
}

KeyFrameDatabase* Atlas::GetKeyFrameDatabase()
{
    return mpKeyFrameDB;
}

void Atlas::SetORBVocabulary(ORBVocabulary* pORBVoc)
{
    mpORBVocabulary = pORBVoc;
}

ORBVocabulary* Atlas::GetORBVocabulary()
{
    return mpORBVocabulary;
}

long unsigned int Atlas::GetNumLivedKF()
{
    unique_lock<mutex> lock(mMutexAtlas);
    long unsigned int num = 0;
    for(Map* pMap_i : mspMaps)
    {
        num += pMap_i->GetAllKeyFrames().size();
    }

    return num;
}

long unsigned int Atlas::GetNumLivedMP() {
    unique_lock<mutex> lock(mMutexAtlas);
    long unsigned int num = 0;
    for (Map* pMap_i : mspMaps) {
        num += pMap_i->GetAllMapPoints().size();
    }

    return num;
}

map<long unsigned int, KeyFrame*> Atlas::GetAtlasKeyframes()
{
    map<long unsigned int, KeyFrame*> mpIdKFs;
    for(Map* pMap_i : mvpBackupMaps)
    {
        vector<KeyFrame*> vpKFs_Mi = pMap_i->GetAllKeyFrames();

        for(KeyFrame* pKF_j_Mi : vpKFs_Mi)
        {
            mpIdKFs[pKF_j_Mi->mnId] = pKF_j_Mi;
        }
    }

    return mpIdKFs;
}

} //namespace ORB_SLAM3
